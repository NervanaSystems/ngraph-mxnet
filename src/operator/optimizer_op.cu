/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file optimizer_op.cu
 * \brief Optimizer operators
 * \author Junyuan Xie
 */
#include "./optimizer_op-inl.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(sgd_update)
.set_attr<FCompute>("FCompute<gpu>", SGDUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", SGDUpdateEx<gpu>);

NNVM_REGISTER_OP(sgd_mom_update)
.set_attr<FCompute>("FCompute<gpu>", SGDMomUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", SGDMomUpdateEx<gpu>);

NNVM_REGISTER_OP(mp_sgd_update)
.set_attr<FCompute>("FCompute<gpu>", MP_SGDUpdate<gpu>);

NNVM_REGISTER_OP(mp_sgd_mom_update)
.set_attr<FCompute>("FCompute<gpu>", MP_SGDMomUpdate<gpu>);

NNVM_REGISTER_OP(adam_update)
.set_attr<FCompute>("FCompute<gpu>", AdamUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", AdamUpdateEx<gpu>);

NNVM_REGISTER_OP(rmsprop_update)
.set_attr<FCompute>("FCompute<gpu>", RMSPropUpdate<gpu>);

NNVM_REGISTER_OP(rmspropalex_update)
.set_attr<FCompute>("FCompute<gpu>", RMSPropAlexUpdate<gpu>);

NNVM_REGISTER_OP(ftrl_update)
.set_attr<FCompute>("FCompute<gpu>", FtrlUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", FtrlUpdateEx<gpu>);

}  // namespace op
}  // namespace mxnet
