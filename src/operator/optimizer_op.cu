/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2016 by Contributors
 * \file optimizer_op.cu
 * \brief Optimizer operators
 * \author Junyuan Xie
 */
#include "./optimizer_op-inl.h"
#include <hipcub/hipcub.hpp>

namespace mxnet {
namespace op {

template<>
void SGDMomStdUpdateDnsRspDnsImpl<gpu>(const SGDMomParam& param,
                                       const OpContext& ctx,
                                       const TBlob& weight,
                                       const NDArray& grad,
                                       const TBlob& mom,
                                       const OpReqType& req,
                                       TBlob *out) {
  using namespace mxnet_op;
  using namespace rowsparse;
  using namespace mshadow;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteInplace) << "kWriteInplace is expected for sparse sgd_mom_update";
  CHECK_GT(weight.shape_.Size(), 0);
  CHECK_GT(mom.shape_.Size(), 0);

  MSHADOW_REAL_TYPE_SWITCH(weight.type_flag_, DType, {
    MSHADOW_IDX_TYPE_SWITCH(grad.aux_type(kIdx), IType, {
      MXNET_ASSIGN_REQ_SWITCH(req, req_type, {
        DType* weight_data = weight.dptr<DType>();
        IType* grad_idx = grad.aux_data(kIdx).dptr<IType>();
        DType* grad_val = grad.data().dptr<DType>();
        DType* mom_data = mom.dptr<DType>();
        DType* out_data = out->dptr<DType>();
        nnvm::dim_t num_rows = weight.shape_[0];
        nnvm::dim_t row_length = weight.shape_.ProdShape(1, weight.ndim());

        nnvm::dim_t* prefix_sum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      Stream<gpu>::GetStream(s));
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
          .get_space_typed<gpu, 1, char>(Shape1(num_rows * sizeof(nnvm::dim_t) +
                                         temp_storage_bytes), s);
        prefix_sum = reinterpret_cast<nnvm::dim_t*>(workspace.dptr_);
        d_temp_storage = workspace.dptr_ + num_rows*sizeof(nnvm::dim_t);
        // mark row flags
        Fill<false>(s, TBlob(prefix_sum, Shape1(num_rows), gpu::kDevMask), kWriteTo, 0);
        if (grad.storage_initialized()) {
          Kernel<MarkRowFlgKernel, gpu>::Launch(s, grad.aux_shape(kIdx)[0],
            prefix_sum, grad_idx);
          // calculate inclusive prefix sum
          hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                        temp_storage_bytes,
                                        prefix_sum,
                                        prefix_sum,
                                        num_rows,
                                        mshadow::Stream<gpu>::GetStream(s));
        }
        Kernel<SGDMomStdDnsRspDnsKernel<req_type>, gpu>::Launch(s, num_rows, row_length,
          out_data, mom_data, weight_data, grad_idx, grad_val, prefix_sum,
          static_cast<DType>(param.clip_gradient), static_cast<DType>(param.momentum),
          static_cast<DType>(param.lr), static_cast<DType>(param.wd),
          static_cast<DType>(param.rescale_grad));
      });
    });
  });
}

template<>
void AdamStdUpdateDnsRspDnsImpl<gpu>(const AdamParam& param,
                                     const OpContext& ctx,
                                     const TBlob& weight,
                                     const NDArray& grad,
                                     const TBlob& mean,
                                     const TBlob& var,
                                     const OpReqType& req,
                                     TBlob *out) {
  using namespace mxnet_op;
  using namespace rowsparse;
  using namespace mshadow;
  Stream<gpu>* s = ctx.get_stream<gpu>();
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteInplace) << "kWriteInplace is expected for sparse adam_update";
  CHECK_GT(weight.shape_.Size(), 0);
  CHECK_GT(mean.shape_.Size(), 0);
  CHECK_GT(var.shape_.Size(), 0);

  MSHADOW_REAL_TYPE_SWITCH(weight.type_flag_, DType, {
    MSHADOW_IDX_TYPE_SWITCH(grad.aux_type(kIdx), IType, {
      MXNET_ASSIGN_REQ_SWITCH(req, req_type, {
        const DType* weight_data = weight.dptr<DType>();
        const IType* grad_idx = grad.aux_data(kIdx).dptr<IType>();
        const DType* grad_val = grad.data().dptr<DType>();
        DType* mean_data = mean.dptr<DType>();
        DType* var_data = var.dptr<DType>();
        DType* out_data = out->dptr<DType>();
        nnvm::dim_t num_rows = weight.shape_[0];
        nnvm::dim_t row_length = weight.shape_.ProdShape(1, weight.ndim());
        nnvm::dim_t* prefix_sum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      Stream<gpu>::GetStream(s));
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
          .get_space_typed<gpu, 1, char>(Shape1(num_rows * sizeof(nnvm::dim_t) +
                                         temp_storage_bytes), s);
        prefix_sum = reinterpret_cast<nnvm::dim_t*>(workspace.dptr_);
        d_temp_storage = workspace.dptr_ + num_rows*sizeof(nnvm::dim_t);
        // mark row flags
        Fill<false>(s, TBlob(prefix_sum, Shape1(num_rows), gpu::kDevMask), kWriteTo, 0);
        if (grad.storage_initialized()) {
          Kernel<MarkRowFlgKernel, gpu>::Launch(s, grad.aux_shape(kIdx)[0],
            prefix_sum, grad_idx);
          // calculate inclusive prefix sum
          hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                        temp_storage_bytes,
                                        prefix_sum,
                                        prefix_sum,
                                        num_rows,
                                        Stream<gpu>::GetStream(s));
        }

        Kernel<AdamStdDnsRspDnsKernel<req_type>, gpu>::Launch(s, num_rows, row_length,
          out_data, mean_data, var_data, weight_data, grad_idx, grad_val, prefix_sum,
          static_cast<DType>(param.clip_gradient), static_cast<DType>(param.beta1),
          static_cast<DType>(param.beta2), static_cast<DType>(param.lr),
          static_cast<DType>(param.wd), static_cast<DType>(param.epsilon),
          static_cast<DType>(param.rescale_grad));
      });
    });
  });
}

NNVM_REGISTER_OP(signsgd_update)
.set_attr<FCompute>("FCompute<gpu>", SignSGDUpdate<gpu>);

NNVM_REGISTER_OP(signum_update)
.set_attr<FCompute>("FCompute<gpu>", SignumUpdate<gpu>);

NNVM_REGISTER_OP(sgd_update)
.set_attr<FCompute>("FCompute<gpu>", SGDUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", SGDUpdateEx<gpu>);

NNVM_REGISTER_OP(sgd_mom_update)
.set_attr<FCompute>("FCompute<gpu>", SGDMomUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", SGDMomUpdateEx<gpu>);

NNVM_REGISTER_OP(mp_sgd_update)
.set_attr<FCompute>("FCompute<gpu>", MP_SGDUpdate<gpu>);

NNVM_REGISTER_OP(mp_sgd_mom_update)
.set_attr<FCompute>("FCompute<gpu>", MP_SGDMomUpdate<gpu>);

NNVM_REGISTER_OP(ftml_update)
.set_attr<FCompute>("FCompute<gpu>", FTMLUpdate<gpu>);

NNVM_REGISTER_OP(adam_update)
.set_attr<FCompute>("FCompute<gpu>", AdamUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", AdamUpdateEx<gpu>);

NNVM_REGISTER_OP(rmsprop_update)
.set_attr<FCompute>("FCompute<gpu>", RMSPropUpdate<gpu>);

NNVM_REGISTER_OP(rmspropalex_update)
.set_attr<FCompute>("FCompute<gpu>", RMSPropAlexUpdate<gpu>);

NNVM_REGISTER_OP(ftrl_update)
.set_attr<FCompute>("FCompute<gpu>", FtrlUpdate<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", FtrlUpdateEx<gpu>);

}  // namespace op
}  // namespace mxnet
