#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2018 by Contributors
 * \file adamw.cu
 * \brief Optimizer operators
 * \author Haibin Lin
 */
#include "./adamw-inl.h"

namespace mxnet {
namespace op {

template<template <typename xpu> class F>
inline void MPUpdateGPU(const nnvm::NodeAttrs& attrs,
                        const OpContext &ctx,
                        const std::vector<TBlob> &inputs,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &outputs) {
  // copy to cpu and check NaN value
  TBlob scale_blob = inputs[inputs.size() - 1];
  MSHADOW_REAL_TYPE_SWITCH(scale_blob.type_flag_, DType, {
    DType scale = 0;
    CUDA_CALL(hipMemcpy(&scale, scale_blob.dptr<DType>(), sizeof(DType),
       hipMemcpyDeviceToHost));
    float scalef = static_cast<float>(scale);
    if (!std::isfinite(scalef) || scalef == 0) return;
    std::vector<TBlob> inputs_wo_scale;
    size_t num_in = inputs.size();
    inputs_wo_scale.reserve(num_in - 1);
    for (size_t i = 0; i < num_in - 1; i++) inputs_wo_scale.emplace_back(inputs[i]);
    F<gpu>::Forward(attrs, ctx, inputs_wo_scale, req, outputs, scalef);
  });
}

NNVM_REGISTER_OP(_adamw_update)
.set_attr<FCompute>("FCompute<gpu>", MPUpdateGPU<AdamWUpdate>);

NNVM_REGISTER_OP(_mp_adamw_update)
.set_attr<FCompute>("FCompute<gpu>", MPUpdateGPU<MPAdamWUpdate>);

}  // namespace op
}  // namespace mxnet
