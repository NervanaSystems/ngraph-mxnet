/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 *  Copyright (c) 2018 by Contributors
 * \file subgraph_op.cu
 * \brief GPU Implementation of subgraph operations
 */

#include "./default_subgraph_op.h"

namespace mxnet {
namespace op {

void DefaultSubgraphOpForward(const OpStatePtr& state_ptr,
                              const OpContext& ctx,
                              const std::vector<NDArray>& inputs,
                              const std::vector<OpReqType>& req,
                              const std::vector<NDArray>& outputs);

NNVM_REGISTER_OP(_default_subgraph_op)
.set_attr<FStatefulComputeEx>("FStatefulComputeEx<gpu>", DefaultSubgraphOpForward);

}  // namespace op
}  // namespace mxnet
