/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file deconvolution.cu
 * \brief
 * \author Wei Wu
*/

#include "./deconvolution-inl.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_deconvolution-inl.h"
#endif  // MXNET_USE_CUDNN

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(DeconvolutionParam param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  // Logic here parallels that in Convolution.cu
  Operator *op = NULL;
  // If 1D deconvolution, use MXNet implementation
  if (param.kernel.ndim() == 1) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new DeconvolutionOp<gpu, DType>(param);
    })
    return op;
  }
#if MXNET_USE_CUDNN == 1
  // On fp16-I/O instances, use fp32 compute (i.e. pseudo-fp16).
  int compute_type = (dtype == mshadow::kFloat16) ? mshadow::kFloat32 : dtype;

  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    if (param.cudnn_off) {
      op = new DeconvolutionOp<gpu, DType>(param);
    } else if (!CuDNNDeconvolutionOp<DType>::Supports(param, compute_type, compute_type, ctx)) {
      LOG(WARNING) <<
        "This deconvolution is not supported by cudnn, MXNET deconvolution is applied.";
      op = new DeconvolutionOp<gpu, DType>(param);
    } else {
      op = new CuDNNDeconvolutionOp<DType>(param, compute_type, compute_type,
                                           *in_shape, *out_shape, ctx);
    }
  })
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new DeconvolutionOp<gpu, DType>(param);
  })
#endif  // MXNET_USE_CUDNN
  return op;
}

}  // namespace op
}  // namespace mxnet
