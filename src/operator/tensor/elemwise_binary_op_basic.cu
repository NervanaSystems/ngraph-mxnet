/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_binary_scalar_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(elemwise_add)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::ComputeWithHalf2<gpu, mshadow::op::plus>);

NNVM_REGISTER_OP(_grad_add)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::ComputeWithHalf2<gpu, mshadow::op::plus>);

NNVM_REGISTER_OP(_backward_add)
.set_attr<FCompute>("FCompute<gpu>",
                    ElemwiseBinaryOp::BackwardUseNoneWithHalf2<gpu, mshadow_op::identity,
                    mshadow_op::identity>);

NNVM_REGISTER_OP(elemwise_sub)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::ComputeWithHalf2<gpu, mshadow::op::minus>);

NNVM_REGISTER_OP(_backward_sub)
.set_attr<FCompute>("FCompute<gpu>",
                    ElemwiseBinaryOp::BackwardUseNoneWithHalf2<gpu, mshadow_op::identity,
                    mshadow_op::negation>);

NNVM_REGISTER_OP(elemwise_mul)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::ComputeWithHalf2<gpu, mshadow::op::mul>);

NNVM_REGISTER_OP(_backward_mul)
.set_attr<FCompute>("FCompute<gpu>",
                    ElemwiseBinaryOp::BackwardUseInWithHalf2<gpu, mshadow_op::right,
                    mshadow_op::left>);

NNVM_REGISTER_OP(elemwise_div)
.set_attr<FCompute>("FCompute<gpu>",
                    ElemwiseBinaryOp::ElemwiseBinaryOp::ComputeWithHalf2<gpu, mshadow::op::div>);

NNVM_REGISTER_OP(_backward_div)
.set_attr<FCompute>("FCompute<gpu>",
                    ElemwiseBinaryOp::BackwardUseInWithHalf2<gpu, mshadow_op::div_grad,
                    mshadow_op::div_rgrad>);

NNVM_REGISTER_OP(_mod)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::ComputeWithHalf2<gpu, mshadow_op::mod>);

NNVM_REGISTER_OP(_backward_mod)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryOp::BackwardUseInWithHalf2<gpu, mshadow_op::mod_grad, mshadow_op::mod_rgrad>);

}  // namespace op
}  // namespace mxnet
