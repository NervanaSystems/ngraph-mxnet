#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file elemwise_unary_op.cu
 * \brief GPU Implementation of unary function.
 */
#include "./elemwise_binary_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(relu)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::KernelCompute<gpu, kernel_launch_op::relu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::KernelComputeEx<gpu, kernel_launch_op::relu>);

NNVM_REGISTER_OP(_backward_relu)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, kernel_launch_op::relu_grad>);

NNVM_REGISTER_OP(sigmoid)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::KernelCompute<gpu, kernel_launch_op::sigmoid>);

NNVM_REGISTER_OP(_backward_sigmoid)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, kernel_launch_op::sigmoid_grad>);

// copy
NNVM_REGISTER_OP(_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::IdentityComputeEx<gpu>);

NNVM_REGISTER_OP(_backward_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(BlockGrad)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

NNVM_REGISTER_OP(make_loss)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

// identity output as first input, but attributes are constrainted to be like rhs
NNVM_REGISTER_OP(_identity_with_attr_like_rhs)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::IdentityComputeFirstItemEx<gpu>);

NNVM_REGISTER_OP(Cast)
.set_attr<FCompute>("FCompute<gpu>", CastCompute<gpu>);

NNVM_REGISTER_OP(_backward_cast)
.set_attr<FCompute>("FCompute<gpu>", CastCompute<gpu>);

// negative
NNVM_REGISTER_OP(negative)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::negation>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::negation>);

// reciprocal
NNVM_REGISTER_OP(reciprocal)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::reciprocal>);

NNVM_REGISTER_OP(_backward_reciprocal)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::reciprocal_grad> >);

// abs
NNVM_REGISTER_OP(abs)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::abs>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::abs>);

NNVM_REGISTER_OP(_backward_abs)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::sign> >);

// sign
NNVM_REGISTER_OP(sign)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sign>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::sign>);

NNVM_REGISTER_OP(_backward_sign)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sign_grad> >);

// round
NNVM_REGISTER_OP(round)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::round>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::round>);

// ceil
NNVM_REGISTER_OP(ceil)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::ceil>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::ceil>);

// floor
NNVM_REGISTER_OP(floor)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::floor>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::floor>);

// trunc
NNVM_REGISTER_OP(trunc)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::trunc>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::trunc>);

// rint
NNVM_REGISTER_OP(rint)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::rint>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::rint>);


// fix
NNVM_REGISTER_OP(fix)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::fix>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::fix>);


// square
NNVM_REGISTER_OP(square)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::square>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::square>);

NNVM_REGISTER_OP(_backward_square)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::square_grad> >);

// sqrt
NNVM_REGISTER_OP(sqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::square_root>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::square_root>);


NNVM_REGISTER_OP(_backward_sqrt)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::square_root_grad> >);

// rsqrt
NNVM_REGISTER_OP(rsqrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::reciprocal_square_root>);

NNVM_REGISTER_OP(_backward_rsqrt)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::reciprocal_square_root_grad> >);

// cbrt
NNVM_REGISTER_OP(cbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::cube_root>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::cube_root>);


NNVM_REGISTER_OP(_backward_cbrt)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::cube_root_grad> >);

// rcbrt
NNVM_REGISTER_OP(rcbrt)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::reciprocal_cube_root>);

NNVM_REGISTER_OP(_backward_rcbrt)
.set_attr<FCompute>("FCompute<gpu>",
  ElemwiseBinaryOp::Compute<gpu, unary_bwd<mshadow_op::reciprocal_cube_root_grad> >);

// exp
NNVM_REGISTER_OP(exp)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::exp>);

// log
NNVM_REGISTER_OP(log)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log>);

// log10
NNVM_REGISTER_OP(log10)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log10>);

// log2
NNVM_REGISTER_OP(log2)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log2>);

NNVM_REGISTER_OP(_backward_log)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::log_grad> >);

// log1p
NNVM_REGISTER_OP(log1p)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log1p>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::log1p>);

NNVM_REGISTER_OP(_backward_log1p)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::log1p_grad> >);

// expm1
NNVM_REGISTER_OP(expm1)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::expm1>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::expm1>);

NNVM_REGISTER_OP(_backward_expm1)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::exp> >);

// sin
NNVM_REGISTER_OP(sin)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sin>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::sin>);

NNVM_REGISTER_OP(_backward_sin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sin_grad> >);

// cos
NNVM_REGISTER_OP(cos)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::cos>);

NNVM_REGISTER_OP(_backward_cos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::cos_grad> >);

// tan
NNVM_REGISTER_OP(tan)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::tan>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::tan>);

NNVM_REGISTER_OP(_backward_tan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::tan_grad> >);

// arcsin
NNVM_REGISTER_OP(arcsin)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arcsin>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arcsin>);

NNVM_REGISTER_OP(_backward_arcsin)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arcsin_grad> >);

// arccos
NNVM_REGISTER_OP(arccos)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arccos>);

NNVM_REGISTER_OP(_backward_arccos)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arccos_grad> >);

// arctan
NNVM_REGISTER_OP(arctan)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arctan>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arctan>);

NNVM_REGISTER_OP(_backward_arctan)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arctan_grad> >);

// degrees
NNVM_REGISTER_OP(degrees)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::degrees>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::degrees>);

NNVM_REGISTER_OP(_backward_degrees)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::degrees_grad> >);

// radians
NNVM_REGISTER_OP(radians)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::radians>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::radians>);

NNVM_REGISTER_OP(_backward_radians)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::radians_grad> >);

// cosh
NNVM_REGISTER_OP(cosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::cosh>);

NNVM_REGISTER_OP(_backward_cosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::cosh_grad> >);

// sinh
NNVM_REGISTER_OP(sinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sinh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::sinh>);

NNVM_REGISTER_OP(_backward_sinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::sinh_grad> >);

// tanh
NNVM_REGISTER_OP(tanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::tanh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::tanh>);

NNVM_REGISTER_OP(_backward_tanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::tanh_grad> >);

// arcsinh
NNVM_REGISTER_OP(arcsinh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arcsinh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arcsinh>);

NNVM_REGISTER_OP(_backward_arcsinh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arcsinh_grad> >);

// arccosh
NNVM_REGISTER_OP(arccosh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arccosh>);

NNVM_REGISTER_OP(_backward_arccosh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arccosh_grad> >);

// arctanh
NNVM_REGISTER_OP(arctanh)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::arctanh>)
.set_attr<FComputeEx>("FComputeEx<gpu>", UnaryOp::ComputeEx<gpu, mshadow_op::arctanh>);

NNVM_REGISTER_OP(_backward_arctanh)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::arctanh_grad> >);

// gamma
NNVM_REGISTER_OP(gamma)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::gamma>);

NNVM_REGISTER_OP(_backward_gamma)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::gamma_grad> >);

// gammaln
NNVM_REGISTER_OP(gammaln)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::gammaln>);

NNVM_REGISTER_OP(_backward_gammaln)
.set_attr<FCompute>("FCompute<gpu>", ElemwiseBinaryOp::Compute<
  gpu, unary_bwd<mshadow_op::gammaln_grad> >);

}  // namespace op
}  // namespace mxnet
