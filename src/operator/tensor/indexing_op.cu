#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2017 by Contributors
 * \file indexing_op.cu
 * \brief
 * \author Siyi Li, Chi Zhang
*/

#include "./indexing_op.h"
#include "./util/tensor_util-inl.cuh"

namespace mxnet {
namespace op {

/*! \brief If there are out-of-bound indices, out will be assigned to 1.
 */

struct is_valid_check {
  template<typename DType>
  MSHADOW_XINLINE static void Map(int i, int32_t* out, const DType* data,
                                  const DType min, const DType max) {
    if (data[i] < min || data[i] > max) *out = 1;
  }
};


struct AddTakeGradRspGPUKernel {
  template<typename DType, typename IType>
  __device__ __forceinline__ static void Map(int tid,
                                             DType* out,
                                             const nnvm::dim_t* prefix_sum,
                                             const IType* data,
                                             const DType* ograd,
                                             const nnvm::dim_t row_length) {
    using nnvm::dim_t;
    const dim_t data_i = tid / row_length;
    const dim_t grad_i = tid % row_length;
    const dim_t irow = static_cast<dim_t>(data[data_i]);
    const dim_t rsp_row = prefix_sum[irow] - 1;
    const DType val = ograd[data_i * row_length + grad_i];
    atomicAdd(static_cast<DType *>(&(out[rsp_row*row_length+grad_i])), val);
  }
};

template<>
void SparseEmbeddingOpForwardRspImpl<gpu>(const OpContext& ctx,
                                          const TBlob& data,
                                          const NDArray& weight,
                                          const OpReqType req,
                                          const TBlob& output) {
  if (req == kNullOp) return;
  using namespace rowsparse;
  using namespace mxnet_op;
  mshadow::Stream<gpu>* s = ctx.get_stream<gpu>();
  // zeros weight
  if (req == kWriteTo && !weight.storage_initialized()) {
    size_t out_size = output.shape_.Size();
    MSHADOW_TYPE_SWITCH(output.type_flag_, DType, {
      Fill<false>(s, TBlob(output.dptr<DType>(), mshadow::Shape1(out_size),
          gpu::kDevMask), kWriteTo, 0);
    })
    return;
  }
  // check out-of-bound indices
  int32_t is_valid = 0;
  MSHADOW_TYPE_SWITCH(data.type_flag_, DType, {
    DType min = 0;
    DType max = static_cast<DType>(weight.shape()[0] - 1);
    DType* data_ptr = data.dptr<DType>();
    size_t data_size = data.shape_.Size();
    Tensor<gpu, 1, char> workspace = ctx.requested[0]
        .get_space_typed<gpu, 1, char>(Shape1(sizeof(int32_t)), s);
    int32_t* is_valid_ptr = reinterpret_cast<int32_t*>(workspace.dptr_);
    Kernel<set_zero, gpu>::Launch(s, 1, is_valid_ptr);
    Kernel<is_valid_check, gpu>::Launch(s, data_size, is_valid_ptr, data_ptr, min, max);
    CUDA_CALL(hipMemcpy(&is_valid, is_valid_ptr, sizeof(int32_t),
              hipMemcpyDeviceToHost));
  })
  CHECK_EQ(is_valid, 0) << "SparseEmbedding input contains data out of bound";
  // the weight is actually dense
  if (weight.aux_shape(kIdx)[0] == weight.shape()[0]) {
    EmbeddingOpForwardDnsImpl<gpu>(s, data, weight.data(), req, output);
  } else {
    EmbeddingOpForwardRspImpl<gpu>(s, data, weight, req, output);
  }
}


template<>
inline void SparseEmbeddingOpBackwardRspImpl<gpu>(const OpContext& ctx,
                                                  const TBlob& ograd,
                                                  const TBlob& data,
                                                  const OpReqType req,
                                                  const NDArray& output) {
  using namespace mshadow;
  using namespace mxnet_op;
  using namespace mshadow::expr;
  using namespace rowsparse;
  using nnvm::dim_t;
  if (req == kNullOp) return;
  CHECK_EQ(req, kWriteTo) << "SparseEmbedding layer doesn't support "
                          << "weight gradient calculation with req != write";

  // Request temporary storage for marking non-zero rows and prefix sum
  Stream<gpu> *s = ctx.get_stream<gpu>();
  dim_t num_rows = output.shape()[0];
  dim_t row_length = output.shape()[1];
  dim_t data_size = static_cast<dim_t>(data.shape_.Size());
  dim_t num_threads;

  MSHADOW_TYPE_SWITCH(data.type_flag_, IType, {
    MSHADOW_SGL_DBL_TYPE_SWITCH(ograd.type_flag_, DType, {
      MSHADOW_IDX_TYPE_SWITCH(output.aux_type(kIdx), RType, {
        dim_t* prefix_sum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      Stream<gpu>::GetStream(s));
        Tensor<gpu, 1, char> workspace = ctx.requested[0]
            .get_space_typed<gpu, 1, char>(Shape1(num_rows * sizeof(dim_t) +
                                           temp_storage_bytes), s);
        prefix_sum = reinterpret_cast<dim_t*>(workspace.dptr_);
        d_temp_storage = workspace.dptr_ + num_rows*sizeof(dim_t);
        num_threads = num_rows;
        Fill<false>(s, TBlob(prefix_sum, Shape1(num_threads), gpu::kDevMask), kWriteTo, 0);
        Kernel<MarkRowFlgKernel, gpu>::Launch(s, data_size, prefix_sum, data.dptr<IType>());

        hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                      temp_storage_bytes,
                                      prefix_sum,
                                      prefix_sum,
                                      num_rows,
                                      mshadow::Stream<gpu>::GetStream(s));
        dim_t nnr = 0;
        CUDA_CALL(hipMemcpy(&nnr, &prefix_sum[num_rows-1], sizeof(dim_t),
            hipMemcpyDeviceToHost));

        if (nnr == 0) {
          FillZerosRspImpl(s, output);
          return;
        }
        output.CheckAndAlloc({Shape1(nnr)});
        RType* grad_row_idx = output.aux_data(kIdx).dptr<RType>();
        // fill row_idx array of output matrix, using the row_flg values
        Kernel<FillRspRowIdxKernel, gpu>::Launch(s, num_rows,
            grad_row_idx, prefix_sum, num_rows);
        // prefill with zeros
        DType* grad_data = output.data().dptr<DType>();
        Fill<false>(s, TBlob(grad_data, Shape1(nnr * row_length), gpu::kDevMask),
            kWriteTo, 0);
        // add the final gradients
        num_threads = row_length * data_size;
        Kernel<AddTakeGradRspGPUKernel, gpu>::Launch(s, num_threads, grad_data, prefix_sum,
            data.dptr<IType>(), ograd.dptr<DType>(), row_length);
      });
    });
  });
}

struct backward_gather_nd_gpu {
  template<typename DType, typename IType>
  MSHADOW_XINLINE static void Map(int i, int N, int M, int K,
                                  const mshadow::Shape<10> strides,
                                  DType* out, const DType* data,
                                  const IType* indices) {
    int offset = 0;
    for (int j = 0; j < M; ++j) {
      offset += strides[j] * static_cast<int>(indices[j*N + i]);
    }
    for (int j = 0; j < K; ++j) {
      atomicAdd(out + (offset + j), data[i * K + j]);
    }
  }
};

template<typename DType, typename IType>
inline void GatherNDBackwardImpl(int N, int M, int K,
                                 const mshadow::Shape<10> strides,
                                 DType* out,
                                 const DType* data,
                                 const IType* indices,
                                 mshadow::Stream<gpu> *s) {
  mxnet_op::Kernel<backward_gather_nd_gpu, gpu>::Launch(s, N, N, M, K, strides, out, data, indices);
}

NNVM_REGISTER_OP(Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpForward<gpu>);

NNVM_REGISTER_OP(_contrib_SparseEmbedding)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpForwardEx<gpu>);

NNVM_REGISTER_OP(_backward_Embedding)
.set_attr<FCompute>("FCompute<gpu>", EmbeddingOpBackward<gpu>);

NNVM_REGISTER_OP(_backward_SparseEmbedding)
.set_attr<FComputeEx>("FComputeEx<gpu>", SparseEmbeddingOpBackwardEx<gpu>);

NNVM_REGISTER_OP(take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpForward<gpu>);

NNVM_REGISTER_OP(_backward_take)
.set_attr<FCompute>("FCompute<gpu>", TakeOpBackward<gpu>);

NNVM_REGISTER_OP(batch_take)
.set_attr<FCompute>("FCompute<gpu>", BatchTakeOpForward<gpu>);

NNVM_REGISTER_OP(one_hot)
.set_attr<FCompute>("FCompute<gpu>", OneHotOpForward<gpu>);

NNVM_REGISTER_OP(gather_nd)
.set_attr<FCompute>("FCompute<gpu>", GatherNDForward<gpu>);

NNVM_REGISTER_OP(scatter_nd)
.set_attr<FCompute>("FCompute<gpu>", ScatterNDForward<gpu>);

NNVM_REGISTER_OP(_backward_gather_nd)
.set_attr<FCompute>("FCompute<gpu>", GatherNDBackward<gpu>);

NNVM_REGISTER_OP(_scatter_set_nd)
.set_attr<FCompute>("FCompute<gpu>", ScatterSetNDForward<gpu>);
}  // namespace op
}  // namespace mxnet
