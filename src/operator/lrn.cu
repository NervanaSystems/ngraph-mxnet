/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file lrn.cu
 * \brief
 * \author Bing Xu
*/

#include "./lrn-inl.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_lrn-inl.h"
#endif

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(LRNParam param, int dtype) {
  Operator *op = NULL;
#if MXNET_USE_CUDNN == 1
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new CuDNNLocalResponseNormOp<DType>(param);
  })
#else
#if CUDA_VERSION == 7000
  LOG(FATAL) << "Due to old CUDA compiler bug, LRN is disabled."
             << "Please upgrade CUDA to 7.5+ or use CUDNN";
#else
  op = new LocalResponseNormOp<gpu>(param);
#endif  // CUDA_VERSION
#endif  // MXNET_USE_CUDNN
  return op;
}

}  // namespace op
}  // namespace mxnet


