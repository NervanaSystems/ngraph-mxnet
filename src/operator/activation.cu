#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/
#include "./activation-inl.h"
#include "./mshadow_op.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_activation-inl.h"
#endif

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(ActivationParam param, int dtype, const TShape& dshape) {
  Operator *op = NULL;
  // SoftReLU not supported by CUDNN yet
  if (param.act_type == activation::kSoftReLU) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new ActivationOp<gpu, mshadow_op::softrelu, mshadow_op::softrelu_grad, DType>();
    })
    return op;
  }

#if MXNET_USE_CUDNN == 1
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new CuDNNActivationOp<DType>(param);
  })
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    switch (param.act_type) {
      case activation::kReLU:
        op = new ActivationOp<gpu, mshadow_op::relu, mshadow_op::relu_grad, DType>();
        break;
      case activation::kSigmoid:
        op = new ActivationOp<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad, DType>();
        break;
      case activation::kTanh:
        op = new ActivationOp<gpu, mshadow_op::tanh, mshadow_op::tanh_grad, DType>();
        break;
      default:
        LOG(FATAL) << "unknown activation";
    }
  })
#endif  // MXNET_USE_CUDNN
  return op;
}
}  // namespace op
}  // namespace mxnet
