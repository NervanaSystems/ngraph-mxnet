/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file convolution.cu
 * \brief
 * \author Bing Xu, Jun Wu
*/

#include "./convolution-inl.h"
#include <vector>
#if MXNET_USE_CUDNN == 1
#include "./cudnn_convolution-inl.h"
#endif  // MXNET_USE_CUDNN

#include "./depthwise_convolution-inl.h"

namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(ConvolutionParam param, int dtype,
                        std::vector<TShape> *in_shape,
                        std::vector<TShape> *out_shape,
                        Context ctx) {
  Operator *op = NULL;
  // If 1D convolution, use MXNet implementation
  if (param.kernel.ndim() == 1) {
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
      op = new ConvolutionOp<gpu, DType>(param);
    })
    return op;
  }

  // depth wise conv
  if (param.num_filter == param.num_group &&
      param.layout.value() == mshadow::kNCHW &&
      param.num_filter == (*in_shape)[conv::kData][1] &&
      param.kernel.ndim() == 2 &&
      param.dilate == mshadow::Shape2(1, 1) &&
      dtype == mshadow::kFloat32) {
    op = new DepthwiseConvolutionOp<float>(param, *in_shape, *out_shape);
    return op;
  }

#if MXNET_USE_CUDNN == 1
  // On fp16-I/O instances, use fp32 compute (i.e. pseudo-fp16).
  int compute_type = (dtype == mshadow::kFloat16) ? mshadow::kFloat32 : dtype;

  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    if (param.cudnn_off) {
      op = new ConvolutionOp<gpu, DType>(param);
    } else if (!CuDNNConvolutionOp<DType>::Supports(param, compute_type, compute_type, ctx)) {
      LOG(WARNING) << "This convolution is not supported by cudnn, MXNET convolution is applied.";
      op = new ConvolutionOp<gpu, DType>(param);
    } else {
      op = new CuDNNConvolutionOp<DType>(param, compute_type, compute_type,
                                         *in_shape, *out_shape, ctx);
    }
  })
#else
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new ConvolutionOp<gpu, DType>(param);
  })
#endif  // MXNET_USE_CUDNN
  return op;
}

}  // namespace op
}  // namespace mxnet

